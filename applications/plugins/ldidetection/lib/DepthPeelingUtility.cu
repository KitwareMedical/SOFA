#include "hip/hip_runtime.h"
//#include <sofa/gpu/cuda/CudaMath.h>
#include <sofa/gpu/cuda/CudaMath.h>

namespace sofa
{

namespace component
{

namespace collision
{
	extern "C"
	{
		void CollisionVolume_count(const void* layers, void* counts, void* count_per_bloc, const int nlayers, const int npixels, const int bsize, bool self);
		void CollisionVolume_write(const void* layers, const void* counts, const void* bloc_input_pos, const void* bloc_output_pos, void* collisions, const int nlayers, const int npixels, const int bsize, const int nblocs, bool self);
	}

__global__ void CollisionVolume_count_kernel(const float4* layers, unsigned int* counts, unsigned int* count_sums, const int nlayers, const int npixels) //, int offset)
{
    //! Dynamically allocated shared memory for gather
    extern  __shared__ unsigned int temp[];
    extern  __shared__ float ftemp[];
    const int index0 = umul24(blockIdx.x,blockDim.x);
    const int index1 = threadIdx.x;
    const int index2 = index1+index1;
    int index = index0+index1;
    //float vol = 0.0f;
    unsigned int count = 0;
    //layers += index;
    layers += index0;
    counts += index;
    //float4 first = *layers;
/*
    int p;
    p = index1;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; //p+=blockDim.x;
    __syncthreads();
    float4 first=make_float4(ftemp[index4+0],ftemp[index4+1],ftemp[index4+2],ftemp[index4+3]);
    __syncthreads();
*/
    float2 f2 = ((const float2*)layers)[index1];
    ftemp[index1] = (index1&1)?f2.x:f2.y;
    f2 = ((const float2*)layers)[index1+blockDim.x];
    ftemp[index1+blockDim.x] = (index1&1)?f2.x:f2.y;
    __syncthreads();
    float2 first=make_float2(ftemp[index2+0],ftemp[index2+1]);
    __syncthreads();
    for(int L=nlayers-2;L>=0;--L)
    {
        layers += npixels;
        //float4 current = *layers;
/*
        p = index1;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; //p+=blockDim.x;
        __syncthreads();
        float4 current=make_float4(ftemp[index4+0],ftemp[index4+1],ftemp[index4+2],ftemp[index4+3]);
        __syncthreads();
*/
        f2 = ((const float2*)layers)[index1];
        ftemp[index1] = (index1&1)?f2.x:f2.y;
        f2 = ((const float2*)layers)[index1+blockDim.x];
        ftemp[index1+blockDim.x] = (index1&1)?f2.x:f2.y;
        __syncthreads();
        float2 current=make_float2(ftemp[index2+0],ftemp[index2+1]);
        __syncthreads();
        //if (current.z!=0.0 && first.z > 0.5 && current.z < 0.5 && floor(first.y*2) != floor(current.y*2))
        if (current.y!=0.0 && first.y > 0.5 && current.y < 0.5 && floor(first.x*2) != floor(current.x*2))
        {
            ++count;
            //vol += current.x - first.x;
        }
        first = current;
    }

    //*counts = count;
    int n = blockDim.x;

    unsigned int v = count;
    for (int i=1;i<n;)
    {
        temp[index1] = v;
        __syncthreads();
        if (index1>=i)
            v += temp[index1 - i];
        i<<=1;
        temp[n + index1] = v;
        __syncthreads();
        if (index1>=i)
            v += temp[n + index1 - i];
        i<<=1;
    }
    *counts = v - count;
    if (index1 == n-1)
        count_sums[blockIdx.x] = v;

    /*
    temp[index1+index1] = (float)count;
    temp[index1+index1+1] = vol;
    __syncthreads();
    vol = temp[index1]+temp[index1+n];
    __syncthreads();
    */
    /*
    while(offset>0)
    {
        if (index1 >= offset && index1 < n)
            temp[index1] = count;
        __syncthreads();
        if (index1+offset < n)
            count += temp[index1+offset];
        n = offset;
        offset >>= 1;
    }
    if (index1 == 0)
        count_sums[blockIdx.x] = count;
    */
}

__global__ void SelfCollisionVolume_count_kernel(const float4* layers, unsigned int* counts, unsigned int* count_sums, const int nlayers, const int npixels) //, int offset)
{
    //! Dynamically allocated shared memory for gather
    extern  __shared__  unsigned int temp[];
    extern  __shared__  float ftemp[];
    const int index0 = umul24(blockIdx.x,blockDim.x);
    const int index1 = threadIdx.x;
    const int index2 = index1+index1;
    int index = index0+index1;
    //float vol = 0.0f;
    unsigned int count = 0;
    //layers += index;
    layers += index0;
    counts += index;
    //float4 first = *layers;
/*
    int p;
    p = index1;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
    ftemp[p] = ((const float*)layers)[p]; //p+=blockDim.x;
    __syncthreads();
    float4 first=make_float4(ftemp[index4+0],ftemp[index4+1],ftemp[index4+2],ftemp[index4+3]);
    __syncthreads();
*/
    float2 f2 = ((const float2*)layers)[index1];
    ftemp[index1] = (index1&1)?f2.x:f2.y;
    f2 = ((const float2*)layers)[index1+blockDim.x];
    ftemp[index1+blockDim.x] = (index1&1)?f2.x:f2.y;
    __syncthreads();
    float first=ftemp[index2+1];
    __syncthreads();

    int inside = floor(2*first)*2-1;
    for(int L=nlayers-2;L>=0;--L)
    {
        layers += npixels;
        //float4 current = *layers;
/*
        p = index1;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; p+=blockDim.x;
        ftemp[p] = ((const float*)layers)[p]; //p+=blockDim.x;
        __syncthreads();
        float4 current=make_float4(ftemp[index4+0],ftemp[index4+1],ftemp[index4+2],ftemp[index4+3]);
        __syncthreads();
*/
        f2 = ((const float2*)layers)[index1];
        ftemp[index1] = (index1&1)?f2.x:f2.y;
        f2 = ((const float2*)layers)[index1+blockDim.x];
        ftemp[index1+blockDim.x] = (index1&1)?f2.x:f2.y;
        __syncthreads();
        float current=ftemp[index2+1];
        __syncthreads();
        if (current > 0.5)
            ++inside;
        else
        {
            --inside;
            if (current!=0.0 && first > 0.5 && inside > 0)
            {
                ++count;
            }
        }
        first = current;
    }

    //*counts = count;
    int n = blockDim.x;

    unsigned int v = count;
    for (int i=1;i<n;)
    {
        temp[index1] = v;
        __syncthreads();
        if (index1>=i)
            v += temp[index1 - i];
        i<<=1;
        temp[n + index1] = v;
        __syncthreads();
        if (index1>=i)
            v += temp[n + index1 - i];
        i<<=1;
    }
    *counts = v - count;
    if (index1 == n-1)
        count_sums[blockIdx.x] = v;
    /*
    temp[index1+index1] = (float)count;
    temp[index1+index1+1] = vol;
    __syncthreads();
    vol = temp[index1]+temp[index1+n];
    __syncthreads();
    */
    /*
    while(offset>0)
    {
        if (index1 >= offset && index1 < n)
            temp[index1] = count;
        __syncthreads();
        if (index1+offset < n)
            count += temp[index1+offset];
        n = offset;
        offset >>= 1;
    }
    if (index1 == 0)
        count_sums[blockIdx.x] = count;
    */
}

void CollisionVolume_count(const void* layers, void* counts, void* count_sums, const int nlayers, const int npixels, const int bsize, bool self)
{
    int nblocs = (npixels+bsize-1)/bsize;
    dim3 threads(bsize,1);
    dim3 grid(nblocs,1);
    /*int offset;
    if (bsize==1)
        offset = 0;
    else
    {
        offset = 1;
        while (offset*2 < bsize)
            offset *= 2;
    }*/
    if (self)
        SelfCollisionVolume_count_kernel<<< grid, threads, 2*bsize * sizeof(unsigned int) >>>((const float4*)layers, (unsigned int*)counts, (unsigned int*)count_sums, nlayers, npixels); //, offset);
    else
        CollisionVolume_count_kernel<<< grid, threads, 2*bsize * sizeof(unsigned int) >>>((const float4*)layers, (unsigned int*)counts, (unsigned int*)count_sums, nlayers, npixels); //, offset);
}

__global__ void CollisionVolume_write_kernel(const float4* layers, const unsigned int* counts, const unsigned int* bloc_input_pos, const unsigned int* bloc_output_pos, float4* collisions, const int nlayers, const int npixels)
{
    //! Dynamically allocated shared memory
    extern  __shared__  unsigned int temp[];
    extern  __shared__  float ftemp[];
    const int index1 = threadIdx.x;
    const int index2 = index1+index1;
    if (index1 == 0)
    {
        temp[0] = bloc_input_pos[blockIdx.x];
        temp[1] = bloc_output_pos[blockIdx.x];
    }
    __syncthreads();
    layers += temp[0]; //+index1;
    collisions += (temp[1] + counts[temp[0]+index1])<<1;
    //float4 first = *layers;
    __syncthreads();
    float2 f2 = ((const float2*)layers)[index1];
    float2 f2b = ((const float2*)layers)[index1+blockDim.x];
    ftemp[index1] = f2.x;
    ftemp[index1+blockDim.x] = f2b.x;
    __syncthreads();
    float4 first;
    first.x = ftemp[index2+0];
    first.z = ftemp[index2+1];
    __syncthreads();
    ftemp[index1] = f2.y;
    ftemp[index1+blockDim.x] = f2b.y;
    __syncthreads();
    first.y = ftemp[index2+0];
    first.w = ftemp[index2+1];
    for(int L=nlayers-2;L>=0;--L)
    {
        layers += npixels;
//        float4 current = *layers;
        __syncthreads();
        f2 = ((const float2*)layers)[index1];
        f2b = ((const float2*)layers)[index1+blockDim.x];
        ftemp[index1] = f2.x;
        ftemp[index1+blockDim.x] = f2b.x;
        __syncthreads();
        float4 current;
        current.x = ftemp[index2+0];
        current.z = ftemp[index2+1];
        __syncthreads();
        ftemp[index1] = f2.y;
        ftemp[index1+blockDim.x] = f2b.y;
        __syncthreads();
        current.y = ftemp[index2+0];
        current.w = ftemp[index2+1];
        if (current.z!=0.0 && first.z > 0.5 && current.z < 0.5 && floor(first.y*2) != floor(current.y*2))
        {
            *collisions = first; ++collisions;
            *collisions = current; ++collisions;
        }
        first = current;
    }
}

__global__ void SelfCollisionVolume_write_kernel(const float4* layers, const unsigned int* counts, const unsigned int* bloc_input_pos, const unsigned int* bloc_output_pos, float4* collisions, const int nlayers, const int npixels)
{
    //! Dynamically allocated shared memory
    extern  __shared__  unsigned int temp[];
    extern  __shared__  float ftemp[];
    const int index1 = threadIdx.x;
    const int index2 = index1+index1;
    if (index1 == 0)
    {
        temp[0] = bloc_input_pos[blockIdx.x];
        temp[1] = bloc_output_pos[blockIdx.x];
    }
    __syncthreads();
    layers += temp[0]; //+index1;
    collisions += (temp[1] + counts[temp[0]+index1])<<1;
    //float4 first = *layers;
    __syncthreads();
    float2 f2 = ((const float2*)layers)[index1];
    float2 f2b = ((const float2*)layers)[index1+blockDim.x];
    ftemp[index1] = f2.x;
    ftemp[index1+blockDim.x] = f2b.x;
    __syncthreads();
    float4 first;
    first.x = ftemp[index2+0];
    first.z = ftemp[index2+1];
    __syncthreads();
    ftemp[index1] = f2.y;
    ftemp[index1+blockDim.x] = f2b.y;
    __syncthreads();
    first.y = ftemp[index2+0];
    first.w = ftemp[index2+1];
    int inside = floor(2*first.z)*2-1;
    for(int L=nlayers-2;L>=0;--L)
    {
        layers += npixels;
//        float4 current = *layers;
        __syncthreads();
        f2 = ((const float2*)layers)[index1];
        f2b = ((const float2*)layers)[index1+blockDim.x];
        ftemp[index1] = f2.x;
        ftemp[index1+blockDim.x] = f2b.x;
        __syncthreads();
        float4 current;
        current.x = ftemp[index2+0];
        current.z = ftemp[index2+1];
        __syncthreads();
        ftemp[index1] = f2.y;
        ftemp[index1+blockDim.x] = f2b.y;
        __syncthreads();
        current.y = ftemp[index2+0];
        current.w = ftemp[index2+1];
        if (current.z > 0.5)
            ++inside;
        else
        {
            --inside;
            if (current.z!=0.0 && first.z > 0.5 && inside > 0)
            {
                *collisions = first; ++collisions;
                *collisions = current; ++collisions;
            }
        }
        first = current;
    }
}

//void CollisionVolume_write(const void* layers, const void* scan_counts, void* collisions, const int nlayers, const int npixels, const int bsize, const int nblocs, bool self)
void CollisionVolume_write(const void* layers, const void* counts, const void* bloc_input_pos, const void* bloc_output_pos, void* collisions, const int nlayers, const int npixels, const int bsize, const int nblocs, bool self)
{
    dim3 threads(bsize,1);
    dim3 grid(nblocs,1);
    if (self)
        SelfCollisionVolume_write_kernel<<< grid, threads, 2*bsize*sizeof(unsigned int) >>>((const float4*)layers, (const unsigned int*)counts, (const unsigned int*)bloc_input_pos, (const unsigned int*)bloc_output_pos, (float4*)collisions, nlayers, npixels);
    else
        CollisionVolume_write_kernel<<< grid, threads, 2*bsize*sizeof(unsigned int) >>>((const float4*)layers, (const unsigned int*)counts, (const unsigned int*)bloc_input_pos, (const unsigned int*)bloc_output_pos, (float4*)collisions, nlayers, npixels);
}

}
}
}
