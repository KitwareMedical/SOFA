#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 RC 1        *
*                (c) 2006-2011 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
/* PART OF THIS FILE IS FROM NVIDIA CUDA SDK particles demo:
 *
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */


#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
#include <sofa/gpu/cuda/mycuda.h>
#include <hip/hip_runtime.h>

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void SpatialGridContainer3f_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x);
    void SpatialGridContainer3f1_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x);
    void SpatialGridContainer_findCellRange(int cellBits, int index0, float cellWidth, int nbPoints, const void* particleHash8, void* cellRange, void* cellGhost);
//void SpatialGridContainer3f_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x);
//void SpatialGridContainer3f1_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x);
}

#define USE_TEX 0

struct GridParams
{
    float cellWidth;
    float invCellWidth;
    int cellMask;
    float halfCellWidth;
    float invHalfCellWidth;
};

// large prime numbers
#define HASH_PX 73856093
#define HASH_PY 19349663
#define HASH_PZ 83492791

//////////////////////
// GPU-side methods //
//////////////////////

#if USE_TEX
texture<int, 1, hipReadModeElementType> cellRangeTex;
#endif

__constant__ GridParams gridParams;

// calculate cell in grid from position
template<class T>
__device__ int3 calcGridPos(T p)
{
    int3 i;
    i.x = __float2int_rd(p.x * gridParams.invCellWidth);
    i.y = __float2int_rd(p.y * gridParams.invCellWidth);
    i.z = __float2int_rd(p.z * gridParams.invCellWidth);
    return i;
}

// calculate address in grid from position
__device__ unsigned int calcGridHashI(int3 p)
{
    //return ((p.x<<10)^(p.y<<5)^(p.z)) & gridParams.cellMask;
    //return ((p.x)^(p.y)^(p.z)) & gridParams.cellMask;
    return (__mul24(HASH_PX,p.x)^__mul24(HASH_PY,p.y)^__mul24(HASH_PZ,p.z)) & gridParams.cellMask;
    //return (p.x) & gridParams.cellMask;
}

// calculate address in grid from position
template<class T>
__device__ unsigned int calcGridHash(T p)
{
    return calcGridHashI(calcGridPos(p));
}


__device__ __inline__ float3 getPos3(const float4* pos, int index0, int index)
{
    float4 p = pos[index];
    return make_float3(p.x,p.y,p.z);
}

__shared__ float ftemp[BSIZE*3];

__device__ __inline__ float3 getPos3(const float3* pos, int index0, int index)
{
    //return pos[index];

    int index03 = __umul24(index0,3);
    int index3 = __umul24(threadIdx.x,3);
    ftemp[threadIdx.x] = ((const float*)pos)[index03+threadIdx.x];
    ftemp[threadIdx.x+BSIZE] = ((const float*)pos)[index03+threadIdx.x+BSIZE];
    ftemp[threadIdx.x+2*BSIZE] = ((const float*)pos)[index03+threadIdx.x+2*BSIZE];
    __syncthreads();
    return make_float3(ftemp[index3],ftemp[index3+1],ftemp[index3+2]);
}

__device__ __inline__ float4 getPos4(const float4* pos, int index0, int index)
{
    return pos[index];
}

__device__ __inline__ float4 getPos4(const float3* pos, int index0, int index)
{
    int index3 = __umul24(threadIdx.x,3);
    pos += index0;
    ftemp[threadIdx.x] = ((const float*)pos)[threadIdx.x];
    ftemp[threadIdx.x+BSIZE] = ((const float*)pos)[threadIdx.x+BSIZE];
    ftemp[threadIdx.x+2*BSIZE] = ((const float*)pos)[threadIdx.x+2*BSIZE];
    __syncthreads();
    return make_float4(ftemp[index3],ftemp[index3+1],ftemp[index3+2],0.0f);
}

__device__ __inline__ float4 getPos4(const float4* pos, int index)
{
    return pos[index];
}

__device__ __inline__ float4 getPos4(const float3* pos, int index)
{
    float3 p = pos[index];
    return make_float4(p.x,p.y,p.z,1.0f);
}

// calculate grid hash value for each particle
template<class TIn>
__global__ void
computeHashD(const TIn* pos,
        unsigned int* particleIndex8, unsigned int*  particleHash8, int n)
{
    int index0 = (blockIdx.x*BSIZE);
    int index = index0 + threadIdx.x;
    int nt = n - index0; if (nt > BSIZE) nt = BSIZE;
    float3 p = getPos3(pos,index0,index);

    int3 hgpos;
    hgpos.x = __float2int_rd(p.x * gridParams.invHalfCellWidth);
    hgpos.y = __float2int_rd(p.y * gridParams.invHalfCellWidth);
    hgpos.z = __float2int_rd(p.z * gridParams.invHalfCellWidth);
    int halfcell = ((hgpos.x&1) + ((hgpos.y&1)<<1) + ((hgpos.z&1)<<2))^7;
    // compute the first cell to be influenced by the particle
    hgpos.x = (hgpos.x-1) >> 1;
    hgpos.y = (hgpos.y-1) >> 1;
    hgpos.z = (hgpos.z-1) >> 1;

    __syncthreads();

    __shared__ int hx[3*BSIZE];
    int x = threadIdx.x;

//    hx[x] = (__mul24(HASH_PX,hgpos.x) << 3)+halfcell;
//    hy[x] = __mul24(HASH_PY,hgpos.y);
//    hz[x] = __mul24(HASH_PZ,hgpos.z);
    hx[x] = ((HASH_PX*hgpos.x) << 3)+halfcell;
    hx[BSIZE+x] = (HASH_PY*hgpos.y);
    hx[2*BSIZE+x] = (HASH_PZ*hgpos.z);
    __syncthreads();
    int3 dH;
    dH.x = (x&1 ? HASH_PX : 0);
    dH.y = (x&2 ? HASH_PY : 0);
    dH.z = (x&4 ? HASH_PZ : 0);
    int x_7 = x&7;
    int index0_8_x_7 = (index0 << 3) + x_7;
    for (unsigned int lx = x>>3; lx < nt; lx+=(BSIZE>>3))
    {
        particleIndex8[index0_8_x_7 + (lx<<3)] = index0 + lx;
        int3 h;
        h.x = hx[lx];
        h.y = hx[BSIZE+lx];
        h.z = hx[2*BSIZE+lx];
        int hc = h.x & 7;
        h.x = (h.x>>3) + dH.x;
        h.y += dH.y;
        h.z += dH.z;
        unsigned int hash = ((h.x ^ h.y ^ h.z) & gridParams.cellMask)<<1;
        if (hc != x_7) ++hash;
        particleHash8[index0_8_x_7 + (lx<<3)] = hash;
    }
}

// find start of each cell in sorted particle list by comparing with previous hash value
// one thread per particle
__global__ void
findCellRangeD(int index0, const unsigned int* particleHash,
        int * cellRange, int* cellGhost, int n)
{
    unsigned int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    __shared__ unsigned int hash[BSIZE];
    if (i < n)
        hash[threadIdx.x] = particleHash[i];

    __syncthreads();

    if (i < n)
    {
        bool firstInCell;
        bool firstGhost;
        unsigned int cur = hash[threadIdx.x];
        if (i == 0)
        {
            firstInCell = true;
            firstGhost = cur&1;
        }
        else
        {
            unsigned int prev;
            if (threadIdx.x > 0)
                prev = hash[threadIdx.x-1];
            else
                prev = particleHash[i-1];
            firstInCell = ((prev>>1) != (cur>>1));
            firstGhost = ((prev != cur) && (cur&1));
            if (firstInCell)
            {
                if ((prev>>1) < (cur>>1)-1)
                    cellRange[ (prev>>1)+1 ] =  (index0+i) | (1U<<31);
                if (!(prev&1)) // no ghost particles in previous cell
                    cellGhost[ prev>>1 ] = index0+i;
            }
        }
        if (firstInCell)
            cellRange[ cur>>1 ] = index0+i;
        if (firstGhost)
            cellGhost[ cur>>1 ] = index0+i;
        if (i == n-1)
        {
            cellRange[ (cur>>1)+1 ] = (index0+n) | (1U<<31);
            if (!(cur&1))
                cellGhost[ cur>>1 ] = index0+n;
        }
    }
}

// rearrange particle data into sorted order
template<class TIn>
__global__ void
reorderDataD(const uint2*  particleHash,  // particle id sorted by hash
        const TIn* oldPos,
        float4* sortedPos, int n
            )
{
    int index0 = __mul24(blockIdx.x, blockDim.x);
    int index = index0 + threadIdx.x;
    if (index < n)
    {
        volatile uint2 sortedData = particleHash[index];
        //float4 pos = getPos4(oldPos,index0,index);
        float4 pos = getPos4(oldPos,sortedData.y);
        sortedPos[index] = pos;
    }
}


//////////////////////
// CPU-side methods //
//////////////////////

void SpatialGridContainer3f_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x)
{
    GridParams p;
    p.cellWidth = cellWidth;
    p.invCellWidth = 1.0f/cellWidth;
    p.cellMask = (1<<cellBits)-1;
    p.halfCellWidth = cellWidth*0.5f;
    p.invHalfCellWidth = 2.0f/cellWidth;
    hipMemcpyToSymbol(HIP_SYMBOL(gridParams), &p, sizeof(GridParams));

    // First compute hash of each particle
    {
        dim3 threads(BSIZE,1);
        dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
        {computeHashD<float3><<< grid, threads >>>((const float3*)x, (unsigned int*)particleIndex8, (unsigned int*)particleHash8, nbPoints); mycudaDebugError("computeHashD<float3>");}
    }
}

void SpatialGridContainer3f1_computeHash(int cellBits, float cellWidth, int nbPoints, void* particleIndex8, void* particleHash8, const void* x)
{
    GridParams p;
    p.cellWidth = cellWidth;
    p.invCellWidth = 1.0f/cellWidth;
    p.cellMask = (1<<cellBits)-1;
    p.halfCellWidth = cellWidth*0.5f;
    p.invHalfCellWidth = 2.0f/cellWidth;
    hipMemcpyToSymbol(HIP_SYMBOL(gridParams), &p, sizeof(GridParams));

    // First compute hash of each particle
    {
        dim3 threads(BSIZE,1);
        dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
        {computeHashD<float4><<< grid, threads >>>((const float4*)x, (unsigned int*)particleIndex8, (unsigned int*)particleHash8, nbPoints); mycudaDebugError("computeHashD<float4>");}
    }
}

void SpatialGridContainer_findCellRange(int cellBits, int index0, float cellWidth, int nbPoints, const void* particleHash8, void* cellRange, void* cellGhost)
{
    hipMemset(cellRange, 0, ((1<<cellBits)+1)*sizeof(int));

    // Then find the start of each cell
    {
        dim3 threads(BSIZE,1);
        dim3 grid((8*nbPoints+BSIZE-1)/BSIZE,1);
        {findCellRangeD<<< grid, threads >>>(index0, (const unsigned int*)particleHash8, (int*)cellRange, (int*)cellGhost, 8*nbPoints); mycudaDebugError("findCellRangeD");}
    }
}
/*
void SpatialGridContainer3f_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x)
{
    dim3 threads(BSIZE,1);
    dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
    {reorderDataD<float3><<< grid, threads >>>((const uint2*)particleHash, (const float3*)x, (float4*)sorted, nbPoints); mycudaDebugError("reorderDataD<float3>");}
}

void SpatialGridContainer3f1_reorderData(int nbPoints, const void* particleHash, void* sorted, const void* x)
{
    dim3 threads(BSIZE,1);
    dim3 grid((nbPoints+BSIZE-1)/BSIZE,1);
    {reorderDataD<float4><<< grid, threads >>>((const uint2*)particleHash, (const float4*)x, (float4*)sorted, nbPoints); mycudaDebugError("reorderDataD<float4>");}
}
*/
#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
