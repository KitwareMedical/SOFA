#include "hip/hip_runtime.h"
/******************************************************************************
*       SOFA, Simulation Open-Framework Architecture, version 1.0 RC 1        *
*                (c) 2006-2011 MGH, INRIA, USTL, UJF, CNRS                    *
*                                                                             *
* This library is free software; you can redistribute it and/or modify it     *
* under the terms of the GNU Lesser General Public License as published by    *
* the Free Software Foundation; either version 2.1 of the License, or (at     *
* your option) any later version.                                             *
*                                                                             *
* This library is distributed in the hope that it will be useful, but WITHOUT *
* ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       *
* FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License *
* for more details.                                                           *
*                                                                             *
* You should have received a copy of the GNU Lesser General Public License    *
* along with this library; if not, write to the Free Software Foundation,     *
* Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301 USA.          *
*******************************************************************************
*                               SOFA :: Modules                               *
*                                                                             *
* Authors: The SOFA Team and external contributors (see Authors.txt)          *
*                                                                             *
* Contact information: contact@sofa-framework.org                             *
******************************************************************************/
#include "CudaCommon.h"
#include "CudaMath.h"
#include "hip/hip_runtime.h"

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
    void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map);
    void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map);
}

struct /*__align__(16)*/ GPUContact
{
    int p1;
    float3 p2;
    float distance;
    float3 normal;
};

struct /*__align__(8)*/ GPUTestEntry
{
    int firstIndex;
    int curSize;
    int maxSize;
    int newIndex;
    int elem1,elem2;
};

//////////////////////
// GPU-side methods //
//////////////////////

__shared__ GPUTestEntry curTestEntry;

__global__ void RigidContactMapperCuda3f_setPoints2_kernel(const GPUTestEntry* tests, const GPUContact* contacts, float3* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = c.p2;
    }
}

__global__ void SubsetContactMapperCuda3f_setPoints1_kernel(unsigned int nbPointsPerElem, const GPUTestEntry* tests, const GPUContact* contacts, int* map)
{
    if (threadIdx.x == 0)
        curTestEntry = tests[blockIdx.x];

    __syncthreads();

    GPUContact c = contacts[curTestEntry.firstIndex + threadIdx.x];
    if (threadIdx.x < curTestEntry.curSize)
    {
        map[curTestEntry.newIndex + threadIdx.x] = umul24(curTestEntry.elem1,nbPointsPerElem) + c.p1;
    }
}


//////////////////////
// CPU-side methods //
//////////////////////

void RigidContactMapperCuda3f_setPoints2(unsigned int size, unsigned int nbTests, unsigned int maxPoints, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    {RigidContactMapperCuda3f_setPoints2_kernel<<< grid, threads >>>((const GPUTestEntry*)tests, (GPUContact*)contacts, (float3*)map); mycudaDebugError("RigidContactMapperCuda3f_setPoints2_kernel");}
}

void SubsetContactMapperCuda3f_setPoints1(unsigned int size, unsigned int nbTests, unsigned int maxPoints, unsigned int nbPointsPerElem, const void* tests, const void* contacts, void* map)
{
    // round up to 16
    //maxPoints = (maxPoints+15)&-16;
    dim3 threads(maxPoints,1);
    dim3 grid(nbTests,1);
    {SubsetContactMapperCuda3f_setPoints1_kernel<<< grid, threads >>>(nbPointsPerElem, (const GPUTestEntry*)tests, (GPUContact*)contacts, (int*)map); mycudaDebugError("SubsetContactMapperCuda3f_setPoints1_kernel");}

}

#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
